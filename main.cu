#include "hip/hip_runtime.h"
#include "benchmark.h"
#include <stdio.h>

__global__ 
void multGPU(vector_t* pvector_in, matrix_t* pmatrix, vector_t* pvector_out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < pmatrix->rows)
    {
        pvector_out->data[i] = 0;
        for(int j = 0; j < pmatrix->collums; j++)
        {
            pvector_out->data[i] += pmatrix->data[(pmatrix->collums * i) + j] * pvector_in->data[j];            
        }
    }   
}

void initalisation(vector_t* &input, matrix_t* &matrix, vector_t* &output);
void clearMemory(vector_t* &input, matrix_t* &matrix, vector_t* &output);

void singleCpuThreadExecution(vector_t* &input, matrix_t* &matrix, vector_t* &output);
void multiCpuThreadExecution(vector_t* &input, matrix_t* &matrix, vector_t* &output);
void gpuExceution(vector_t* &input, matrix_t* &matrix, vector_t* &output);
void printError(vector_t* &vector, float should, const char* const name);

int main()
{    
    matrix_t* matrix;
    vector_t* input;
    vector_t* output;

    initalisation(input, matrix, output);

    gpuExceution(input, matrix, output);
    printError(output, OUTPUT_LENGTH, "GPU");
    multiCpuThreadExecution(input, matrix, output);  
    printError(output, OUTPUT_LENGTH, "CPU multi thread");
    singleCpuThreadExecution(input, matrix, output);
    printError(output, OUTPUT_LENGTH, "CPU single thread");
    clearMemory(input, matrix, output);
     
    return 0;
}

void initalisation(vector_t* &input, matrix_t* &matrix, vector_t* &output)
{
    matrix = initMatrixOnCPU(OUTPUT_LENGTH, INPUT_LENGTH);
    input = initVectorOnCPU(INPUT_LENGTH);
    output = initVectorOnCPU(OUTPUT_LENGTH);

    setMatrixValues(matrix);
}

void clearMemory(vector_t* &input, matrix_t* &matrix, vector_t* &output)
{
    deleteMatrixOnCPU(matrix);
    deleteVectorOnCPU(input);
    deleteVectorOnCPU(output);   
}

void singleCpuThreadExecution(vector_t* &input, matrix_t* &matrix, vector_t* &output)
{
    setInputValues(input);
    clock_t start = clock();
    multSingleThreadCPU(input, matrix, output);
    clock_t fin = clock();

    printf("%.3Fms singel Thread CPU time\n",1000. * (double)(fin - start) / (double)CLOCKS_PER_SEC);
}

void multiCpuThreadExecution(vector_t* &input, matrix_t* &matrix, vector_t* &output)
{
    setInputValues(input);
    clock_t start = clock();
    multCPU(input, matrix, output);
    clock_t fin = clock();

    printf("%.3Fms CPU time with %d threads\n", 1000. * (double)(fin - start) / (double)CLOCKS_PER_SEC, THREADS);
}

void gpuExceution(vector_t* &input, matrix_t* &matrix, vector_t* &output)
{
    setInputValues(input);
    matrix = moveMatrixToGPU(matrix);
    input = moveVectorToGPU(input);
    output = moveVectorToGPU(output);

    clock_t start = clock();
    multGPU<<<1, OUTPUT_LENGTH>>>(input, matrix, output);
    clock_t fin = clock();
    
    matrix = moveMatrixToCPU(matrix);
    input = moveVectorToCPU(input);
    output = moveVectorToCPU(output);

    printf("%.3Fms GPU time\n",1000. * (double)(fin - start) / (double)CLOCKS_PER_SEC);
}

void printError(vector_t* &vector, float should, const char* const name)
{
    float error = 0;
    for (int i = 0; i < vector->collums; i++)
    {
        error = max(error, abs(vector->data[i] - should));
    }
    
    printf("Max %s error: %f\n",name, error);
}